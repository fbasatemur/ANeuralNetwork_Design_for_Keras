#include "hip/hip_runtime.h"
#include ""
#include "CpuGpuMat.h"
#include "KernelSigmoid.cuh"
#include <math.h>


__global__ void gpuSigmoidActivation(float* GpuP, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		GpuP[id] = (float)(1.0 / (1.0 + exp(-1.0 * (double)GpuP[id])));
	}
}


void gpuSigmoid(CpuGpuMat* Mat) {

	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(Mat->Size) / double(threadsPerBlock));

	gpuSigmoidActivation << < blocksPerGrid, threadsPerBlock >> > ((float*)Mat->GpuP, Mat->Size);
}
