#include "hip/hip_runtime.h"
#include ""
#include "CpuGpuMat.h"
#include "KernelDense.cuh"
#include <math.h>


__global__ void gpuMatrixMult(float* gpuMat1, float* gpuMat2, float* gpuMat3, int m1Rows, int m1Cols, int m2Cols, int inStartIndex, int resStartIndex)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0;

	if (row < m1Rows && col < m2Cols) {
		for (int i = 0; i < m1Cols; i++) {

			sum += gpuMat1[inStartIndex + row * m1Cols + i] * gpuMat2[i * m2Cols + col];
		}
		gpuMat3[resStartIndex + row * m2Cols + col] = sum;
	}
}


void gpuMatrixMultiplication(CpuGpuMat* Mat1, CpuGpuMat* Mat2, CpuGpuMat* Mat3, int inStartIndex, int resStartIndex)
{
	int threadsPerBlock = 32;

	int gridCols = ceil(double(Mat2->Cols) / double(threadsPerBlock));
	int gridRows = ceil(double(Mat1->Rows) / double(threadsPerBlock));

	dim3 gridDim(gridCols, gridRows);
	dim3 blockDim(threadsPerBlock, threadsPerBlock);

	gpuMatrixMult << < gridDim, blockDim >> > ((float*)Mat1->GpuP, (float*)Mat2->GpuP, (float*)Mat3->GpuP, Mat1->Rows, Mat1->Cols, Mat2->Cols, inStartIndex, resStartIndex);
}
