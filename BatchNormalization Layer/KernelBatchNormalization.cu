#include "hip/hip_runtime.h"

#include ""
#include "CpuGpuMat.h"
#include "KernelBatchNormalization.cuh"
#include <math.h>


__global__ void gpuBatchNorm(float* gpuResult, float* gpuBeta, float* gpuGamma, float* gpuMovingMean, float* gpuMovingVar, float epsilon, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		gpuResult[id] = (gpuResult[id] - gpuMovingMean[id]) / sqrt(gpuMovingVar[id] + epsilon) * gpuGamma[id] + gpuBeta[id];
	}
}


void gpuBatchNormalization(CpuGpuMat* result, CpuGpuMat* beta, CpuGpuMat* gamma, CpuGpuMat* movingMean, CpuGpuMat* movingVariance, float epsilon) {

	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(beta->Size) / double(threadsPerBlock));

	gpuBatchNorm << < blocksPerGrid, threadsPerBlock >> > ((float*)result->GpuP, (float*)beta->GpuP, (float*)gamma->GpuP, (float*)movingMean->GpuP, (float*)movingVariance->GpuP, epsilon, beta->Size);
}