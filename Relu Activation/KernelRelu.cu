#include "hip/hip_runtime.h"
#include ""
#include "CpuGpuMat.h"
#include "KernelRelu.cuh"
#include <math.h>


__global__ void gpuReluActivation(float* GpuP, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		GpuP[id] = GpuP[id] > 0 ? GpuP[id] : 0;
	}
}


void gpuRelu(CpuGpuMat* Mat)
{
	int threadsPerBlock = 32;
	int blocksPerGrid = ceil(double(Mat->Size) / double(threadsPerBlock));

	gpuReluActivation << < blocksPerGrid, threadsPerBlock >> > ((float*)Mat->GpuP, Mat->Size);
}